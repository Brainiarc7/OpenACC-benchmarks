#include <cstdlib>
#include <iostream>
#include <sstream>

using namespace std;

int main(int argc, char* argv[])
{
	if (argc < 2)
	{
		cout << "Usage: " << argv[0] << " <program>" << endl;
		return 0;
	}

	// Check if zero rank is dedicated to data collection only.
	int cpu_only_master = 0;
	char* cmaster = getenv("CPU_ONLY_MASTER");
	if (cmaster)
		cpu_only_master = atoi(cmaster);
	if (cpu_only_master)
		cpu_only_master = 1;

	// Get the number of available GPUs.
	int count = 0;
	hipError_t err = hipGetDeviceCount(&count);
	if (err != hipSuccess)
	{
		cerr << "Error in hipGetDeviceCount: " <<
			hipGetErrorString(err) << endl;
		return 1;
	}

	// Get the MPI world size.
	char* csize = getenv("OMPI_COMM_WORLD_SIZE");
	if (!csize)
	{
		cerr << "Cannot determine the MPI world size. Are you using OpenMPI?" << endl;
		return 1;
	}

	// Check MPI world size does not exceed the
	// number of available GPUs.
	int size = atoi(csize);
	if (size - cpu_only_master > count)
	{
		cerr << "MPI world size exceeds the number of available GPUs" << endl;
		return 1;
	}

	// Get the MPI process rank.
	char* crank = getenv("OMPI_COMM_WORLD_RANK");
	if (!crank)
	{
		cerr << "Cannot determine the MPI process rank. Are you using OpenMPI?" << endl;
		return 1;
	}

	// Reset device to delete the currenly used CUDA context.
	err = hipDeviceReset();
	if (err != hipSuccess)
	{
		cerr << "Error in hipDeviceReset: " << hipGetErrorString(err) << endl;
		return 1;
	}

	// In CPU_ONLY_MASTER mode - switch the master node to
	// the CPU runmode.
	int rank = atoi(crank);
	if (cpu_only_master)
	{
		if (rank == 0)
		{
			const char* zero = "0";
			setenv("kernelgen_runmode", zero, 1);
		}
	}

	// Execute entire MPI process with the only one GPU visible,
	// which index is either the same as the MPI process rank or
	// less by 1, depending on the master node mode.
	// XXX: Note this mapping does not account the case of
	// multi-head cluster, where each node has several GPUs.
	// In this case one needs to mod rank by the number of available
	// GPUs.
	if (cpu_only_master && (rank > 0))
	{
		rank--;
		stringstream strrank;
		strrank << rank;
		string srank = strrank.str();
		setenv("CUDA_VISIBLE_DEVICES", srank.c_str(), 1);
	}
	else
		setenv("CUDA_VISIBLE_DEVICES", crank, 1);
	execv(argv[1], argv + 1); 

	return 0;
}

